#include "hip/hip_runtime.h"
#include "gpu.h"

#include <algorithm>
#include <hip/hip_runtime.h>

__constant__ WorkNum gpu_powers[8192];
__device__ unsigned int solution_count = 0;

inline __device__ int
gpu_binary_search(const IndexNum elements_count,
                  const WorkNum search) {
  IndexNum l = 0;
  IndexNum r = elements_count - 1;
  IndexNum m;
  while (l <= r) {
    m = (l + r) / 2;
    if (search < gpu_powers[m])
      r = m - 1;
    else if (search > gpu_powers[m])
      l = m + 1;
    else
     return l;
  }
  return -1;
}

__global__ void
AntonCrechetovGPUKernel(const IndexNum elements_count, Solution* solutions) {
  const int x0 = blockIdx.x + 1;
  const int x1 = threadIdx.x + blockIdx.y * blockDim.x + 1;
  if (x1 <= x0)
    return;
  for (int x2 = x1 + 1; x2 < elements_count; ++x2) {
    for (int x3 = x2 + 1; x3 < elements_count; ++x3) {
      const WorkNum sum = gpu_powers[x0] + gpu_powers[x1] + gpu_powers[x2] + gpu_powers[x3];
      auto s = gpu_binary_search(elements_count, sum);
      if (s > 0) {
        size_t indx = atomicInc(&solution_count, 1);
        solutions[indx].a = x0;
        solutions[indx].b = x1;
        solutions[indx].c = x2;
        solutions[indx].d = x3;
        solutions[indx].e = s;
        //printf("%d %d %d %d %d\n", x0, x1, x2, x3, s);
      }
    }
  }
}

__global__ void
NaiveGPUKernel(const IndexNum elements_count, Solution* solutions) {
  const int x0 = blockIdx.x + 1;
  const int x1 = threadIdx.x + blockIdx.y * blockDim.x + 1;
  if (x1 >= x0)
    return;
  const WorkNum s1 = gpu_powers[x0] + gpu_powers[x1];
  for (int x2 = 1; x2 < x1; ++x2) {
    const WorkNum s2 = s1 + gpu_powers[x2];
    for (int x3 = 1; x3 < x2; ++x3) {
      const WorkNum sum = s2 + gpu_powers[x3];
      auto s = gpu_binary_search(elements_count, sum);
      if (s > 0) {
        size_t indx = atomicInc(&solution_count, 1);
        solutions[indx].a = x0;
        solutions[indx].b = x1;
        solutions[indx].c = x2;
        solutions[indx].d = x3;
        solutions[indx].e = s;
        //printf("%d %d %d %d %d\n", x0, x1, x2, x3, s);
      }
    }
  }
}

__global__ void
FirstOptimizedGPUKernel(const IndexNum elements_count, Solution* solutions) {
  const int x0 = blockIdx.x + 1;
  const int x1 = threadIdx.x + blockIdx.y * blockDim.x + 1;
  if (x1 >= x0)
    return;
  IndexNum rs = 5;
  for (int x2 = 1; x2 < x1; ++x2) {
    const WorkNum s2 = gpu_powers[x0] + gpu_powers[x1] + gpu_powers[x2];
    while (rs > 0 && gpu_powers[rs] > s2)
      --rs;
    for (int x3 = 1; x3 <x2; ++x3) {
      const WorkNum sum = s2 + gpu_powers[x3];
      while (rs < elements_count - 1 && gpu_powers[rs] < sum)
        ++rs;
      auto s = gpu_binary_search(elements_count, sum);
      if (s > 0) {
        size_t indx = atomicInc(&solution_count, 1);
        solutions[indx].a = x0;
        solutions[indx].b = x1;
        solutions[indx].c = x2;
        solutions[indx].d = x3;
        solutions[indx].e = s;
        //printf("%d %d %d %d %d\n", x0, x1, x2, x3, s);
      }
    }
  }
}

__global__ void
SecondOptimizedGPUKernel(const IndexNum elements_count, Solution* solutions) {
  const int x0 = blockIdx.x + 1;
  const int x1 = threadIdx.x + blockIdx.y * blockDim.x + 1;
  if (x1 >= x0)
    return;
  IndexNum rs = 5;
  const WorkNum s1 = gpu_powers[x0] + gpu_powers[x1];
  for (int x2 = 1; x2 < x1; ++x2) {
    const WorkNum s2 = s1 + gpu_powers[x2];
    while (rs > 0 && gpu_powers[rs] > s2)
      --rs;
    for (IndexNum x3 = 1; x3 < x2; x3++) {
      // go straight to the first appropriate x3, mod 30
      if (WorkNum err30 = (x0 + x1 + x2 + x3 - rs) % 30)
        x3 += 30 - err30;
      if (x3 >= x2)
        break;
      auto sum = s2 + gpu_powers[x3];
      while (rs < elements_count - 1 && gpu_powers[rs] < sum)
        ++rs;
      if (gpu_powers[rs] == sum) {
        size_t indx = atomicInc(&solution_count, 1);
        solutions[indx].a = x0;
        solutions[indx].b = x1;
        solutions[indx].c = x2;
        solutions[indx].d = x3;
        solutions[indx].e = rs;
        //printf("%d %d %d %d %d\n", x0, x1, x2, x3, rs);
      }
    }
  }
}

template<typename F>
std::vector<Solution>
ComputeOnGpu(const IndexNum elements_count, const std::vector<WorkNum>& powers, F kernel) {
  std::vector<Solution> result;
  hipDeviceSynchronize();
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_powers),
                                       powers.data(),
                                       sizeof(WorkNum) * elements_count);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to hipMemcpyToSymbol (error code %s)!\n", hipGetErrorString(err));
    return result;
  }

  // Allocate the device output vector for solutions
  Solution *device_solutions = nullptr;
  size_t solutions_size = sizeof(Solution) * 100;
  err = hipMalloc(&device_solutions, solutions_size);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to allocate device vector for solutions (error code %s)!\n", 
            hipGetErrorString(err));
    return result;
  }

  // Launch CUDA Kernel
  int blocks_x = elements_count;
  int threads = std::min(1024, static_cast<int>(elements_count));
  int blocks_y = (elements_count + threads - 1) / threads;
  kernel(elements_count,
         blocks_x,
         blocks_y,
         threads,
         device_solutions);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
    return result;
  }
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to wait kernel (error code %s)!\n", hipGetErrorString(err));
    return result;
  }
  return result;
}

std::vector<Solution>
AntonCrechetovGPU(const IndexNum elements_count, const std::vector<WorkNum>& powers,
                  const IndexNum from, const IndexNum to) {
  auto kernel = [](int elements_count,
                   int blocks_x,
                   int blocks_y,
                   int threads,
                   Solution *device_solutions) {
    dim3 grid(blocks_x, blocks_y);
    AntonCrechetovGPUKernel<<<grid, threads>>>(
        elements_count, device_solutions);
  };
  return ComputeOnGpu(elements_count, powers, kernel);
}

std::vector<Solution>
NaiveGPU(const IndexNum elements_count, const std::vector<WorkNum>& powers,
         const IndexNum from, const IndexNum to) {
  auto kernel = [](int elements_count,
                   int blocks_x,
                   int blocks_y,
                   int threads,
                   Solution *device_solutions) {
    dim3 grid(blocks_x, blocks_y);
    NaiveGPUKernel<<<grid, threads>>>(
        elements_count, device_solutions);
  };
  return ComputeOnGpu(elements_count, powers, kernel);
}

std::vector<Solution>
FirstOptimizedGPU(const IndexNum elements_count, const std::vector<WorkNum>& powers,
                  const IndexNum from, const IndexNum to) {
  auto kernel = [](int elements_count,
                   int blocks_x,
                   int blocks_y,
                   int threads,
                   Solution *device_solutions) {
    dim3 grid(blocks_x, blocks_y);
    FirstOptimizedGPUKernel<<<grid, threads>>>(
      elements_count, device_solutions);
  };
  return ComputeOnGpu(elements_count, powers, kernel);
}

std::vector<Solution>
SecondOptimizedGPU(const IndexNum elements_count, const std::vector<WorkNum>& powers,
                   const IndexNum from, const IndexNum to) {
  auto kernel = [](int elements_count,
                   int blocks_x,
                   int blocks_y,
                   int threads,
                   Solution *device_solutions) {
    dim3 grid(blocks_x, blocks_y);
    SecondOptimizedGPUKernel<<<grid, threads>>>(
      elements_count, device_solutions);
  };
  return ComputeOnGpu(elements_count, powers, kernel);
}
